#include <hipcub/hipcub.hpp>

using namespace hipcub;

template <typename T> void cubMax(const T *d_in, T *d_out, int64_t num_items) {
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

  CubDebugExit(DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));

  // Allocate temporary storage
  CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  // Compute Sum
  CubDebugExit(DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));

  if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
}

extern "C" {
void cubMax_int32(const int32_t *d_in, int32_t *d_out, int64_t num_items) {
  cubMax(d_in, d_out, num_items);
}

void cubMax_int64(const int64_t *d_in, int64_t *d_out, int64_t num_items) {
  cubMax(d_in, d_out, num_items);
}

void cubMax_float(const float *d_in, float *d_out, int64_t num_items) {
  cubMax(d_in, d_out, num_items);
}

void cubMax_double(const double *d_in, double *d_out, int64_t num_items) {
  cubMax(d_in, d_out, num_items);
}
}
