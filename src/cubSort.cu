#include <hipcub/hipcub.hpp>
#include <stdio.h>

using namespace hipcub;

template <typename T> void cubSortPairs(const T *d_keys_in, T *d_keys_out, const int64_t *d_values_in, int64_t *d_values_out, size_t N) {
  size_t temp_storage_bytes = 0;
  void *d_temp_storage = NULL;

  CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory
  // run SortPairs once to determine the necessary size of d_temp_storage
  CubDebugExit(DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out, N));
  CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  CubDebugExit(DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out, N));
  if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
}

extern "C" {
void cubSortPairs_int32(const int32_t *d_keys_in, int32_t *d_keys_out, const int64_t *d_values_in, int64_t *d_values_out, size_t N) {
  cubSortPairs(d_keys_in, d_keys_out, d_values_in, d_values_out, N);
}

void cubSortPairs_int64(const int32_t *d_keys_in, int32_t *d_keys_out, const int64_t *d_values_in, int64_t *d_values_out, size_t N) {
  cubSortPairs(d_keys_in, d_keys_out, d_values_in, d_values_out, N);
}

void cubSortPairs_float(const float *d_keys_in, float *d_keys_out, const int64_t *d_values_in, int64_t *d_values_out, size_t N) {
  cubSortPairs(d_keys_in, d_keys_out, d_values_in, d_values_out, N);
}

void cubSortPairs_double(const double *d_keys_in, double *d_keys_out, const int64_t *d_values_in, int64_t *d_values_out, size_t N) {
  cubSortPairs(d_keys_in, d_keys_out, d_values_in, d_values_out, N);
}
}
