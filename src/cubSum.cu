#include <hipcub/hipcub.hpp>
#include <stdio.h>
#include "hip_error_check.h"

using namespace hipcub;

#define DebugExit(x) if (HipcubDebug(x)) exit(1);

template <typename T> void cubSum(const T *d_in, T *d_out, int64_t num_items) {
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

  DebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));

  // Allocate temporary storage
  //CudaSafeCall(hipMalloc(&d_temp_storage, temp_storage_bytes));
  DebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  // Compute Sum
  DebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));

  //CudaSafeCall(hipFree(d_temp_storage));
  if (d_temp_storage) DebugExit(g_allocator.DeviceFree(d_temp_storage));
}

extern "C" {
void cubSum_int32(const int32_t *d_in, int32_t *d_out, int64_t num_items) {
  cubSum(d_in, d_out, num_items);
}

void cubSum_int64(const int64_t *d_in, int64_t *d_out, int64_t num_items) {
  cubSum(d_in, d_out, num_items);
}

void cubSum_float(const float *d_in, float *d_out, int64_t num_items) {
  cubSum(d_in, d_out, num_items);
}

void cubSum_double(const double *d_in, double *d_out, int64_t num_items) {
  cubSum(d_in, d_out, num_items);
}
}
